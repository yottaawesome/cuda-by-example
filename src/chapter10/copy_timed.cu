/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

#define SIZE    (64*1024*1024)


float cuda_malloc_test( int size, bool up ) {
    hipEvent_t     start, stop;
    int             *a, *dev_a;
    float           elapsedTime;

    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    a = (int*)malloc( size * sizeof( *a ) );
    HANDLE_NULL( a );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              size * sizeof( *dev_a ) ) );

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    for (int i=0; i<100; i++) {
        if (up)
            HANDLE_ERROR( hipMemcpy( dev_a, a,
                                  size * sizeof( *dev_a ),
                                  hipMemcpyHostToDevice ) );
        else
            HANDLE_ERROR( hipMemcpy( a, dev_a,
                                  size * sizeof( *dev_a ),
                                  hipMemcpyDeviceToHost ) );
    }
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    free( a );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    return elapsedTime;
}


float cuda_host_alloc_test( int size, bool up ) {
    hipEvent_t     start, stop;
    int             *a, *dev_a;
    float           elapsedTime;

    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    HANDLE_ERROR( hipHostAlloc( (void**)&a,
                                 size * sizeof( *a ),
                                 hipHostMallocDefault ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              size * sizeof( *dev_a ) ) );

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    for (int i=0; i<100; i++) {
        if (up)
            HANDLE_ERROR( hipMemcpy( dev_a, a,
                                  size * sizeof( *a ),
                                  hipMemcpyHostToDevice ) );
        else
            HANDLE_ERROR( hipMemcpy( a, dev_a,
                                  size * sizeof( *a ),
                                  hipMemcpyDeviceToHost ) );
    }
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    HANDLE_ERROR( hipHostFree( a ) );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    return elapsedTime;
}


int main( void ) {
    float           elapsedTime;
    float           MB = (float)100*SIZE*sizeof(int)/1024/1024;


    // try it with hipMalloc
    elapsedTime = cuda_malloc_test( SIZE, true );
    printf( "Time using hipMalloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy up:  %3.1f\n",
            MB/(elapsedTime/1000) );

    elapsedTime = cuda_malloc_test( SIZE, false );
    printf( "Time using hipMalloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy down:  %3.1f\n",
            MB/(elapsedTime/1000) );

    // now try it with hipHostAlloc
    elapsedTime = cuda_host_alloc_test( SIZE, true );
    printf( "Time using hipHostAlloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy up:  %3.1f\n",
            MB/(elapsedTime/1000) );

    elapsedTime = cuda_host_alloc_test( SIZE, false );
    printf( "Time using hipHostAlloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy down:  %3.1f\n",
            MB/(elapsedTime/1000) );
}
