#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)


__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}
        

int main( void ) {
    hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // initialize the stream
    HANDLE_ERROR( hipStreamCreate( &stream ) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c,
                              N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    HANDLE_ERROR( hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N) {
        // copy the locked memory to the device, async
        HANDLE_ERROR( hipMemcpyAsync( dev_a, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_b, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream ) );

        kernel<<<N/256,256,0,stream>>>( dev_a, dev_b, dev_c );

        // copy the data from device to locked memory
        HANDLE_ERROR( hipMemcpyAsync( host_c+i, dev_c,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream ) );

    }
    // copy result chunk from locked to full buffer
    HANDLE_ERROR( hipStreamSynchronize( stream ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    HANDLE_ERROR( hipHostFree( host_a ) );
    HANDLE_ERROR( hipHostFree( host_b ) );
    HANDLE_ERROR( hipHostFree( host_c ) );
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    HANDLE_ERROR( hipStreamDestroy( stream ) );

    return 0;
}

