#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d/10.0f - ticks/7.0f) /
                                         (d/10.0f + 1.0f));    
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255;
}

struct DataBlock {
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame( DataBlock *d, int ticks ) {
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<blocks,threads>>>( d->dev_bitmap, ticks );

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );
}

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

int main( void ) {
    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                            (void (*)(void*))cleanup );
}
