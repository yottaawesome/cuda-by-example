#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    __shared__ float    shared[16][16];

    // now calculate the value at that position
    const float period = 128.0f;

    shared[threadIdx.x][threadIdx.y] =
            255 * (sinf(x*2.0f*PI/ period) + 1.0f) *
                  (sinf(y*2.0f*PI/ period) + 1.0f) / 4.0f;

    // removing this syncthreads shows graphically what happens
    // when it doesn't exist.  this is an example of why we need it.
    __syncthreads();

    ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = shared[15-threadIdx.x][15-threadIdx.y];
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap,
                              bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;

    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( dev_bitmap );

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );
                              
    HANDLE_ERROR( hipFree( dev_bitmap ) );
                              
    bitmap.display_and_exit();
}


