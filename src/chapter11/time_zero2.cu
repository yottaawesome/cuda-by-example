#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid =
            imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );


__global__ void dot( int size, float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


float malloc_test( int size ) {
    hipEvent_t     start, stop;
    float           *a, *b, c, *partial_c;
    float           *dev_a, *dev_b, *dev_partial_c;
    float           elapsedTime;

    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // allocate memory on the CPU side
    a = (float*)malloc( size*sizeof(float) );
    b = (float*)malloc( size*sizeof(float) );
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              size*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              size*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c,
                              blocksPerGrid*sizeof(float) ) );

    // fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, size*sizeof(float),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, size*sizeof(float),
                              hipMemcpyHostToDevice ) ); 

    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_c, dev_partial_c,
                              blocksPerGrid*sizeof(float),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_partial_c ) );

    // free memory on the CPU side
    free( a );
    free( b );
    free( partial_c );

    // free events
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    printf( "Value calculated:  %f\n", c );

    return elapsedTime;
}


float cuda_host_alloc_test( int size ) {
    hipEvent_t     start, stop;
    float           *a, *b, c, *partial_c;
    float           *dev_a, *dev_b, *dev_partial_c;
    float           elapsedTime;

    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // allocate the memory on the CPU
    HANDLE_ERROR( hipHostAlloc( (void**)&a,
                              size*sizeof(float),
                              hipHostMallocWriteCombined |
                                     hipHostMallocMapped ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&b,
                              size*sizeof(float),
                              hipHostMallocWriteCombined |
                                     hipHostMallocMapped ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&partial_c,
                              blocksPerGrid*sizeof(float),
                              hipHostMallocMapped ) );

    // find out the GPU pointers
    HANDLE_ERROR( hipHostGetDevicePointer( &dev_a, a, 0 ) );
    HANDLE_ERROR( hipHostGetDevicePointer( &dev_b, b, 0 ) );
    HANDLE_ERROR( hipHostGetDevicePointer( &dev_partial_c,
                                            partial_c, 0 ) );

    // fill in the host memory with data
    for (int i=0; i<size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );

    HANDLE_ERROR( hipDeviceSynchronize() );
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    HANDLE_ERROR( hipHostFree( a ) );
    HANDLE_ERROR( hipHostFree( b ) );
    HANDLE_ERROR( hipHostFree( partial_c ) );

    // free events
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );

    printf( "Value calculated:  %f\n", c );

    return elapsedTime;
}


int main( void ) {
    hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (prop.canMapHostMemory != 1) {
        printf( "Device can not map memory.\n" );
        return 0;
    }

    float           elapsedTime;

    HANDLE_ERROR( hipSetDeviceFlags( hipDeviceMapHost ) );

    // try it with malloc
    elapsedTime = malloc_test( N );
    printf( "Time using hipMalloc:  %3.1f ms\n",
            elapsedTime );

    // now try it with hipHostAlloc
    elapsedTime = cuda_host_alloc_test( N );
    printf( "Time using hipHostAlloc:  %3.1f ms\n",
            elapsedTime );
}
