#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"


#define imin(a,b) (a<b?a:b)

#define     N    (33*1024*1024)
const int threadsPerBlock = 256;
const int blocksPerGrid =
            imin( 32, (N/2+threadsPerBlock-1) / threadsPerBlock );


__global__ void dot( int size, float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


struct DataStruct {
    int     deviceID;
    int     size;
    int     offset;
    float   *a;
    float   *b;
    float   returnValue;
};


void* routine( void *pvoidData ) {
    DataStruct  *data = (DataStruct*)pvoidData;
    if (data->deviceID != 0) {
        HANDLE_ERROR( hipSetDevice( data->deviceID ) );
        HANDLE_ERROR( hipSetDeviceFlags( hipDeviceMapHost ) );
    }

    int     size = data->size;
    float   *a, *b, c, *partial_c;
    float   *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the CPU side
    a = data->a;
    b = data->b;
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipHostGetDevicePointer( &dev_a, a, 0 ) );
    HANDLE_ERROR( hipHostGetDevicePointer( &dev_b, b, 0 ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c,
                              blocksPerGrid*sizeof(float) ) );

    // offset 'a' and 'b' to where this GPU is gets it data
    dev_a += data->offset;
    dev_b += data->offset;

    dot<<<blocksPerGrid,threadsPerBlock>>>( size, dev_a, dev_b,
                                            dev_partial_c );
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_c, dev_partial_c,
                              blocksPerGrid*sizeof(float),
                              hipMemcpyDeviceToHost ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }

    HANDLE_ERROR( hipFree( dev_partial_c ) );

    // free memory on the CPU side
    free( partial_c );

    data->returnValue = c;
    return 0;
}


int main( void ) {
    int deviceCount;
    HANDLE_ERROR( hipGetDeviceCount( &deviceCount ) );
    if (deviceCount < 2) {
        printf( "We need at least two compute 1.0 or greater "
                "devices, but only found %d\n", deviceCount );
        return 0;
    }

    hipDeviceProp_t  prop;
    for (int i=0; i<2; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        if (prop.canMapHostMemory != 1) {
            printf( "Device %d can not map memory.\n", i );
            return 0;
        }
    }

    float *a, *b;
    HANDLE_ERROR( hipSetDevice( 0 ) );
    HANDLE_ERROR( hipSetDeviceFlags( hipDeviceMapHost ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&a, N*sizeof(float),
                              hipHostMallocWriteCombined |
                              hipHostMallocPortable |
                              hipHostMallocMapped ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&b, N*sizeof(float),
                              hipHostMallocWriteCombined |
                              hipHostMallocPortable      |
                              hipHostMallocMapped ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // prepare for multithread
    DataStruct  data[2];
    data[0].deviceID = 0;
    data[0].offset = 0;
    data[0].size = N/2;
    data[0].a = a;
    data[0].b = b;

    data[1].deviceID = 1;
    data[1].offset = N/2;
    data[1].size = N/2;
    data[1].a = a;
    data[1].b = b;

    CUTThread   thread = start_thread( routine, &(data[1]) );
    routine( &(data[0]) );
    end_thread( thread );


    // free memory on the CPU side
    HANDLE_ERROR( hipHostFree( a ) );
    HANDLE_ERROR( hipHostFree( b ) );

    printf( "Value calculated:  %f\n",
            data[0].returnValue + data[1].returnValue );

    return 0;
}

