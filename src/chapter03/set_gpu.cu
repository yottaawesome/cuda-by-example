/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

int main( void ) {
    hipDeviceProp_t  prop;
    int dev;

    HANDLE_ERROR( hipGetDevice( &dev ) );
    printf( "ID of current CUDA device:  %d\n", dev );

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    HANDLE_ERROR( hipSetDevice( dev ) );
}
