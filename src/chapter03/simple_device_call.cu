#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

__device__ int addem( int a, int b ) {
    return a + b;
}

__global__ void add( int a, int b, int *c ) {
    *c = addem( a, b );
}

int main( void ) {
    int c;
    int *dev_c;
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1,1>>>( 2, 7, dev_c );

    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}


