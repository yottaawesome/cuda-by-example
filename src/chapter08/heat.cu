#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "../common/gpu_anim.h"

#define DIM 1024
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
texture<float>  texConstSrc;
texture<float>  texIn;
texture<float>  texOut;



// this kernel takes in a 2-d array of floats
// it updates the value-of-interest by a scaled value based
// on itself and its nearest neighbors
__global__ void blend_kernel( float *dst,
                              bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0)   left++;
    if (x == DIM-1) right--; 

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0)   top += DIM;
    if (y == DIM-1) bottom -= DIM;

    float   t, l, c, r, b;
    if (dstOut) {
        t = tex1Dfetch(texIn,top);
        l = tex1Dfetch(texIn,left);
        c = tex1Dfetch(texIn,offset);
        r = tex1Dfetch(texIn,right);
        b = tex1Dfetch(texIn,bottom);

    } else {
        t = tex1Dfetch(texOut,top);
        l = tex1Dfetch(texOut,left);
        c = tex1Dfetch(texOut,offset);
        r = tex1Dfetch(texOut,right);
        b = tex1Dfetch(texOut,bottom);
    }
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

// NOTE - texOffsetConstSrc could either be passed as a
// parameter to this function, or passed in __constant__ memory
// if we declared it as a global above, it would be
// a parameter here: 
// __global__ void copy_const_kernel( float *iptr,
//                                    size_t texOffset )
__global__ void copy_const_kernel( float *iptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex1Dfetch(texConstSrc,offset);
    if (c != 0)
        iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

void anim_gpu( uchar4* outputBitmap, DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( outputBitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    HANDLE_ERROR( hipUnbindTexture( texIn ) );
    HANDLE_ERROR( hipUnbindTexture( texOut ) );
    HANDLE_ERROR( hipUnbindTexture( texConstSrc ) );
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    GPUAnimBitmap bitmap( DIM, DIM, &data );
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap.image_size();

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_constSrc,
                              imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texConstSrc,
                                   data.dev_constSrc,
                                   imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texIn,
                                   data.dev_inSrc,
                                   imageSize ) );

    HANDLE_ERROR( hipBindTexture( NULL, texOut,
                                   data.dev_outSrc,
                                   imageSize ) );

    // intialize the constant data
    float *temp = (float*)malloc( imageSize );
    for (int i=0; i<DIM*DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x>300) && (x<600) && (y>310) && (y<601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700+100] = MIN_TEMP;
    temp[DIM*300+300] = MIN_TEMP;
    temp[DIM*200+700] = MIN_TEMP;
    for (int y=800; y<900; y++) {
        for (int x=400; x<500; x++) {
            temp[x+y*DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );    

    // initialize the input data
    for (int y=800; y<DIM; y++) {
        for (int x=0; x<200; x++) {
            temp[x+y*DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );

    bitmap.anim_and_exit( (void (*)(uchar4*,void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}
