#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "../common/gpu_anim.h"

#define DIM 1024

__global__ void kernel( uchar4 *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d/10.0f - ticks/7.0f) /
                                         (d/10.0f + 1.0f));    
    ptr[offset].x = grey;
    ptr[offset].y = grey;
    ptr[offset].z = grey;
    ptr[offset].w = 255;
}

void generate_frame( uchar4 *pixels, void*, int ticks ) {
    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( pixels, ticks );
}

int main( void ) {
    GPUAnimBitmap  bitmap( DIM, DIM, NULL );

    bitmap.anim_and_exit(
        (void (*)(uchar4*,void*,int))generate_frame, NULL );
}
