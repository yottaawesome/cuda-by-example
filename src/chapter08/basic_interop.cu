#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"


PFNGLBINDBUFFERARBPROC    glBindBuffer     = NULL;
PFNGLDELETEBUFFERSARBPROC glDeleteBuffers  = NULL;
PFNGLGENBUFFERSARBPROC    glGenBuffers     = NULL;
PFNGLBUFFERDATAARBPROC    glBufferData     = NULL;

#define     DIM    512

GLuint  bufferObj;
hipGraphicsResource *resource;

// based on ripple code, but uses uchar4 which is the type of data
// graphic inter op uses. see screenshot - basic2.png
__global__ void kernel( uchar4 *ptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x/(float)DIM - 0.5f;
    float fy = y/(float)DIM - 0.5f;
    unsigned char   green = 128 + 127 *
                            sin( abs(fx*100) - abs(fy*100) );

    // accessing uchar4 vs unsigned char*
    ptr[offset].x = 0;
    ptr[offset].y = green;
    ptr[offset].z = 0;
    ptr[offset].w = 255;
}

static void key_func( unsigned char key, int x, int y ) {
    switch (key) {
        case 27:
            // clean up OpenGL and CUDA
            HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) );
            glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );
            glDeleteBuffers( 1, &bufferObj );
            exit(0);
    }
}

static void draw_func( void ) {
    // we pass zero as the last parameter, because out bufferObj is now
    // the source, and the field switches from being a pointer to a
    // bitmap to now mean an offset into a bitmap object
    glDrawPixels( DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0 );
    glutSwapBuffers();
}


int main( int argc, char **argv ) {
    hipDeviceProp_t  prop;
    int dev;

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 0;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );

    // tell CUDA which dev we will be using for graphic interop
    // from the programming guide:  Interoperability with OpenGL
    //     requires that the CUDA device be specified by
    //     cudaGLSetGLDevice() before any other runtime calls.

    HANDLE_ERROR( cudaGLSetGLDevice( dev ) );

    // these GLUT calls need to be made before the other OpenGL
    // calls, else we get a seg fault
    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
    glutInitWindowSize( DIM, DIM );
    glutCreateWindow( "bitmap" );

    glBindBuffer    = (PFNGLBINDBUFFERARBPROC)GET_PROC_ADDRESS("glBindBuffer");
    glDeleteBuffers = (PFNGLDELETEBUFFERSARBPROC)GET_PROC_ADDRESS("glDeleteBuffers");
    glGenBuffers    = (PFNGLGENBUFFERSARBPROC)GET_PROC_ADDRESS("glGenBuffers");
    glBufferData    = (PFNGLBUFFERDATAARBPROC)GET_PROC_ADDRESS("glBufferData");

    // the first three are standard OpenGL, the 4th is the CUDA reg 
    // of the bitmap these calls exist starting in OpenGL 1.5
    glGenBuffers( 1, &bufferObj );
    glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
    glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4,
                  NULL, GL_DYNAMIC_DRAW_ARB );

    HANDLE_ERROR( 
        hipGraphicsGLRegisterBuffer( &resource, 
                                      bufferObj, 
                                      cudaGraphicsMapFlagsNone ) );

    // do work with the memory dst being on the GPU, gotten via mapping
    HANDLE_ERROR( hipGraphicsMapResources( 1, &resource, NULL ) );
    uchar4* devPtr;
    size_t  size;
    HANDLE_ERROR( 
        hipGraphicsResourceGetMappedPointer( (void**)&devPtr, 
                                              &size, 
                                              resource) );

    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( devPtr );
    HANDLE_ERROR( hipGraphicsUnmapResources( 1, &resource, NULL ) );

    // set up GLUT and kick off main loop
    glutKeyboardFunc( key_func );
    glutDisplayFunc( draw_func );
    glutMainLoop();
}
