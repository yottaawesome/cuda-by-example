#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

#define SIZE    (100*1024*1024)

int main( void ) {
    unsigned char *buffer =
                     (unsigned char*)big_random_block( SIZE );

    // capture the start time
    clock_t         start, stop;
    start = clock();

    unsigned int    histo[256];
    for (int i=0; i<256; i++)
        histo[i] = 0;

    for (int i=0; i<SIZE; i++)
        histo[buffer[i]]++;

    stop = clock();
    float   elapsedTime = (float)(stop - start) /
                          (float)CLOCKS_PER_SEC * 1000.0f;
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    long histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum:  %ld\n", histoCount );

    free( buffer );
    return 0;
}
