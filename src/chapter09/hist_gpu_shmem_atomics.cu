#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

#define SIZE    (100*1024*1024)


__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {

    // clear out the accumulation buffer called temp
    // since we are launched with 256 threads, it is easy
    // to clear that memory with one write per thread
    __shared__  unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    // calculate the starting index and the offset to the next
    // block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd( &temp[buffer[i]], 1 );
        i += stride;
    }
    // sync the data from the above writes to shared memory
    // then add the shared memory values to the values from
    // the other thread blocks using global memory
    // atomic adds
    // same as before, since we have 256 threads, updating the
    // global histogram is just one write per thread!
    __syncthreads();
    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

int main( void ) {
    unsigned char *buffer =
                     (unsigned char*)big_random_block( SIZE );

    // capture the start time
    // starting the timer here so that we include the cost of
    // all of the operations on the GPU.  if the data were
    // already on the GPU and we just timed the kernel
    // the timing would drop from 74 ms to 15 ms.  Very fast.
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    // allocate memory on the GPU for the file's data
    unsigned char *dev_buffer;
    unsigned int *dev_histo;
    HANDLE_ERROR( hipMalloc( (void**)&dev_buffer, SIZE ) );
    HANDLE_ERROR( hipMemcpy( dev_buffer, buffer, SIZE,
                              hipMemcpyHostToDevice ) );

    HANDLE_ERROR( hipMalloc( (void**)&dev_histo,
                              256 * sizeof( int ) ) );
    HANDLE_ERROR( hipMemset( dev_histo, 0,
                              256 * sizeof( int ) ) );

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t  prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2,256>>>( dev_buffer,
                                    SIZE, dev_histo );
    
    unsigned int    histo[256];
    HANDLE_ERROR( hipMemcpy( histo, dev_histo,
                              256 * sizeof( int ),
                              hipMemcpyDeviceToHost ) );

    // get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    long histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum:  %ld\n", histoCount );

    // verify that we have the same counts via CPU
    for (int i=0; i<SIZE; i++)
        histo[buffer[i]]--;
    for (int i=0; i<256; i++) {
        if (histo[i] != 0)
            printf( "Failure at %d!\n", i );
    }

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    hipFree( dev_histo );
    hipFree( dev_buffer );
    free( buffer );
    return 0;
}
