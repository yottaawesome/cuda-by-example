#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "lock.h"

#define SIZE    (100*1024*1024)
#define ELEMENTS    (SIZE / sizeof(unsigned int))
#define HASH_ENTRIES     1024


struct Entry {
    unsigned int    key;
    void            *value;
    Entry           *next;
};

struct Table {
    size_t  count;
    Entry   **entries;
    Entry   *pool;
};


__device__ __host__ size_t hash( unsigned int key,
                                 size_t count ) {
    return key % count;
}

void initialize_table( Table &table, int entries,
                       int elements ) {
    table.count = entries;
    HANDLE_ERROR( hipMalloc( (void**)&table.entries,
                              entries * sizeof(Entry*)) );
    HANDLE_ERROR( hipMemset( table.entries, 0,
                              entries * sizeof(Entry*) ) );
    HANDLE_ERROR( hipMalloc( (void**)&table.pool,
                               elements * sizeof(Entry)) );
}

void copy_table_to_host( const Table &table, Table &hostTable) {
    hostTable.count = table.count;
    hostTable.entries = (Entry**)calloc( table.count,
                                         sizeof(Entry*) );
    hostTable.pool = (Entry*)malloc( ELEMENTS *
                                     sizeof( Entry ) );

    HANDLE_ERROR( hipMemcpy( hostTable.entries, table.entries,
                              table.count * sizeof(Entry*),
                              hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( hostTable.pool, table.pool,
                              ELEMENTS * sizeof( Entry ),
                              hipMemcpyDeviceToHost ) );

    for (int i=0; i<table.count; i++) {
        if (hostTable.entries[i] != NULL)
            hostTable.entries[i] =
                (Entry*)((size_t)hostTable.entries[i] -
                (size_t)table.pool + (size_t)hostTable.pool);
    }
    for (int i=0; i<ELEMENTS; i++) {
        if (hostTable.pool[i].next != NULL)
            hostTable.pool[i].next =
                (Entry*)((size_t)hostTable.pool[i].next -
                (size_t)table.pool + (size_t)hostTable.pool);
    }
}

void free_table( Table &table ) {
    HANDLE_ERROR( hipFree( table.pool ) );
    HANDLE_ERROR( hipFree( table.entries ) );
}

__global__ void add_to_table( unsigned int *keys, void **values, 
                              Table table, Lock *lock ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid < ELEMENTS) {
        unsigned int key = keys[tid];
        size_t hashValue = hash( key, table.count );
        for (int i=0; i<32; i++) {
            if ((tid % 32) == i) {
                Entry *location = &(table.pool[tid]);
                location->key = key;
                location->value = values[tid];
                lock[hashValue].lock();
                location->next = table.entries[hashValue];
                table.entries[hashValue] = location;
                lock[hashValue].unlock();
            }
        }
        tid += stride;
    }
}

void verify_table( const Table &dev_table ) {
    Table   table;
    copy_table_to_host( dev_table, table );

    int count = 0;
    for (size_t i=0; i<table.count; i++) {
        Entry   *current = table.entries[i];
        while (current != NULL) {
            ++count;
            if (hash( current->key, table.count ) != i)
                printf( "%d hashed to %ld, but was located at %ld\n",
                        current->key,
                        hash(current->key, table.count), i );
            current = current->next;
        }
    }
    if (count != ELEMENTS)
        printf( "%d elements found in hash table.  Should be %ld\n",
                count, ELEMENTS );
    else
        printf( "All %d elements found in hash table.\n", count );

    free( table.pool );
    free( table.entries );
}


int main( void ) {
    unsigned int *buffer =
                     (unsigned int*)big_random_block( SIZE );

    unsigned int *dev_keys;
    void         **dev_values;
    HANDLE_ERROR( hipMalloc( (void**)&dev_keys, SIZE ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_values, SIZE ) );
    HANDLE_ERROR( hipMemcpy( dev_keys, buffer, SIZE,
                              hipMemcpyHostToDevice ) );
    // copy the values to dev_values here
    // filled in by user of this code example

    Table table;
    initialize_table( table, HASH_ENTRIES, ELEMENTS );

    Lock    lock[HASH_ENTRIES];
    Lock    *dev_lock;
    HANDLE_ERROR( hipMalloc( (void**)&dev_lock,
                              HASH_ENTRIES * sizeof( Lock ) ) );
    HANDLE_ERROR( hipMemcpy( dev_lock, lock,
                              HASH_ENTRIES * sizeof( Lock ),
                              hipMemcpyHostToDevice ) );

    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    add_to_table<<<60,256>>>( dev_keys, dev_values,
                              table, dev_lock );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to hash:  %3.1f ms\n", elapsedTime );

    verify_table( table );

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    free_table( table );
    HANDLE_ERROR( hipFree( dev_lock ) );
    HANDLE_ERROR( hipFree( dev_keys ) );
    HANDLE_ERROR( hipFree( dev_values ) );
    free( buffer );
    return 0;
}

