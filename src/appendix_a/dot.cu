#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "lock.h"


// this example is the dot from a previous chapter, modified so that it
// doesn't have to do the final step on the CPU.  Each grid block has
// to add its float value to the global total, but since we do not have
// an atomicAddFloat, we can do it this way


#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid =
            imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );

__global__ void dot( Lock lock, float *a,
                     float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        // wait until we get the lock
        lock.lock();
       // we have the lock at this point, update and release
        *c += cache[0];
        lock.unlock();
    }
}


int main( void ) {
    float   *a, *b, c = 0;
    float   *dev_a, *dev_b, *dev_c;

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c,
                              sizeof(float) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(float),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N*sizeof(float),
                              hipMemcpyHostToDevice ) ); 
    HANDLE_ERROR( hipMemcpy( dev_c, &c, sizeof(float),
                              hipMemcpyHostToDevice ) ); 

    Lock    lock;
    dot<<<blocksPerGrid,threadsPerBlock>>>( lock, dev_a,
                                            dev_b, dev_c );

    // copy c back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( &c, dev_c,
                              sizeof(float),
                              hipMemcpyDeviceToHost ) );

    #define sum_squares(x)  (x*(x+1)*(2*x+1)/6)
    printf( "Does GPU value %.6g = %.6g?\n", c,
             2 * sum_squares( (float)(N - 1) ) );

    // free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free memory on the cpu side
    free( a );
    free( b );
}
