#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
texture<float,2>  texConstSrc;
texture<float,2>  texIn;
texture<float,2>  texOut;

__global__ void blend_kernel( float *dst,
                              bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b;
    if (dstOut) {
        t = tex2D(texIn,x,y-1);
        l = tex2D(texIn,x-1,y);
        c = tex2D(texIn,x,y);
        r = tex2D(texIn,x+1,y);
        b = tex2D(texIn,x,y+1);
    } else {
        t = tex2D(texOut,x,y-1);
        l = tex2D(texOut,x-1,y);
        c = tex2D(texOut,x,y);
        r = tex2D(texOut,x+1,y);
        b = tex2D(texOut,x,y+1);
    }
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void copy_const_kernel( float *iptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex2D(texConstSrc,x,y);
    if (c != 0)
        iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;
    CPUAnimBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

void anim_gpu( DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i=0; i<90; i++) {
        float   *in, *out;
        if (dstOut) {
            in  = d->dev_inSrc;
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
            in  = d->dev_outSrc;
        }
        copy_const_kernel<<<blocks,threads>>>( in );
        blend_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );
    hipUnbindTexture( texConstSrc );
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    CPUAnimBitmap bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_constSrc,
                              imageSize ) );

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    HANDLE_ERROR( hipBindTexture2D( NULL, texConstSrc,
                                   data.dev_constSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texIn,
                                   data.dev_inSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texOut,
                                   data.dev_outSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    // initialize the constant data
    float *temp = (float*)malloc( imageSize );
    for (int i=0; i<DIM*DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x>300) && (x<600) && (y>310) && (y<601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700+100] = MIN_TEMP;
    temp[DIM*300+300] = MIN_TEMP;
    temp[DIM*200+700] = MIN_TEMP;
    for (int y=800; y<900; y++) {
        for (int x=400; x<500; x++) {
            temp[x+y*DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );    

    // initialize the input data
    for (int y=800; y<DIM; y++) {
        for (int x=0; x<200; x++) {
            temp[x+y*DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );

    bitmap.anim_and_exit( (void (*)(void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}

